#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

#define TILE 16 

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
			             int numARows, int numAColumns,
			             int numBRows, int numBColumns,
			             int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    //@@ You have to use shared memory for this MP
	
	__shared__ float SA[TILE][TILE];
	__shared__ float SB[TILE][TILE];
	
	int bx = blockIdx.x ; int by = blockIdx.y ;
	int tx = threadIdx.x; int ty = threadIdx.y; 
	
	int r =  by * TILE + ty;
	int c =  bx * TILE + tx;
	
	float pVal = 0.0; 

		for(int i = 0; i < (TILE + numAColumns-1)/TILE ; ++i)
		{
			if(r < numARows && i*TILE+ tx < numAColumns)				
				SA[ty][tx] = A[r * numAColumns +i*TILE+ tx];
			else SA[ty][tx] = 0.0;
			
			if((i*TILE + ty) < numBRows && c < numBColumns)
				SB[ty][tx] = B[(i*TILE + ty)*numBColumns + c];
			else SB[ty][tx] = 0.0;
			
			__syncthreads();
		
			for(int k = 0; k < TILE; ++k)
			{
				pVal += SA[ty][k]*SB[k][tx];
			}
			__syncthreads();
		}
		
	  if (r < numCRows && c < numCColumns)
		  C[((by * blockDim.y + threadIdx.y)*numCColumns)+(bx*blockDim.x)+tx] = pVal;
	
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
	hostC = (float*)malloc(numCRows*numCColumns*sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	int sizeA = numAColumns * numARows * sizeof(float);
    int sizeB = numBColumns * numBRows * sizeof(float);
	int sizeC = numCColumns * numCRows * sizeof(float);
	hipMalloc((void**)&deviceA,sizeA);
	hipMalloc((void**)&deviceB,sizeB);
	hipMalloc((void**)&deviceC,sizeC);
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	hipMemcpy(deviceA, hostA,sizeA, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB,sizeB, hipMemcpyHostToDevice);
	hipMemcpy(deviceC, hostC,sizeC, hipMemcpyHostToDevice);
	
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
     dim3 dimBlock(16,16,1);
	 dim3 dimGrid((numCColumns+dimBlock.x-1)/dimBlock.x,(numCRows+ dimBlock.y -1)/dimBlock.y,1);	

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
	matrixMultiply<<<dimGrid,dimBlock>>>(deviceA,deviceB, deviceC,numARows, numAColumns,numBRows, numBColumns,numCRows,numCColumns);
   
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostC,deviceC,sizeC,hipMemcpyDeviceToHost);
	
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}